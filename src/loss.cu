#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <vector>
#include "HugeCTR/include/loss.hpp"
#include "HugeCTR/include/utils.cuh"

namespace HugeCTR {

Loss::Loss(const std::shared_ptr<const Tensor<float>> &label_tensor,
           const std::shared_ptr<Tensor<float>> &input_tensor,
           const std::shared_ptr<Tensor<float>> &loss_tensor,
           const std::shared_ptr<Regularizer> regularizer, int device_id, int total_gpu_count,
           float scaler)
    : label_tensors_(1, label_tensor),
      input_tensors_(1, input_tensor),
      loss_tensors_(1, loss_tensor),
      regularizer_(regularizer),
      device_id_(device_id),
      total_gpu_count_(total_gpu_count),
      scaler_(scaler) {}

void Loss::fused_loss_computation(hipStream_t stream) {
  CudaDeviceContext context(get_device_id());

  const auto &input_tensor = input_tensors_[0];
  const auto &label_tensor = label_tensors_[0];
  const auto &loss_tensor = loss_tensors_[0];

  const auto &input_dim = input_tensor->get_dims();
  const auto &label_dim = label_tensor->get_dims();

  bool row_major = (input_tensor->get_format() == TensorFormat_t::HW);

  int batch_size = row_major ? input_dim[0] : input_dim[1];
  int feature_dim = row_major ? input_dim[1] : input_dim[0];

  float *input = input_tensor->get_ptr();
  const float *label = label_tensor->get_ptr();
  float *loss = loss_tensor->get_ptr();

  float rterm = 0.0f;
  if (regularizer_) {
    regularizer_->compute_rterm(stream);
    rterm = regularizer_->get_rterm();
  }

  do_fused_loss_computation(input, label, loss, batch_size, feature_dim, scaler_, rterm, stream);

  if (regularizer_) {
    regularizer_->initialize_wgrad(stream);
  }

#ifndef NDEBUG
  hipDeviceSynchronize();
  CK_CUDA_THROW_(hipGetLastError());
#endif
}

CrossEntropyLoss::CrossEntropyLoss(const std::shared_ptr<const Tensor<float>> &label_tensor,
                                   const std::shared_ptr<Tensor<float>> &input_tensor,
                                   const std::shared_ptr<Tensor<float>> &loss_tensor,
                                   const std::shared_ptr<Regularizer> regularizer, int device_id,
                                   int total_gpu_count, float scaler)
    : Loss(label_tensor, input_tensor, loss_tensor, regularizer, device_id, total_gpu_count,
           scaler) {
  if (input_tensor->get_format() != label_tensor->get_format())
    CK_THROW_(Error_t::WrongInput, "Format of input tensor and label tensor don't match");

  const auto &input_dim = input_tensor->get_dims();
  const auto &label_dim = label_tensor->get_dims();
  bool row_major = (input_tensor->get_format() == TensorFormat_t::HW);
  int feature_dim = row_major ? input_dim[1] : input_dim[0];

  if (feature_dim != 2)
    CK_THROW_(Error_t::WrongInput, "The feature dimension of CE loss input should be 2");
  if (row_major && input_dim[0] != label_dim[0])
    CK_THROW_(Error_t::WrongInput, "The batch sizes of input tensor and label tensor are not same");
  if (!row_major && input_dim[1] != label_dim[1])
    CK_THROW_(Error_t::WrongInput, "The batch sizes of input tensor and label tensor are not same");
}

// Suppose we use one thread to calculate one sample
__global__ void CrossEntropy_Kernel(float *input, const float *label, float *cel_loss,
                                    int batch_size, int total_gpu_count, int feature_dim,
                                    bool row_major, float scaler, float rterm) {
  int tid = threadIdx.x;
  extern __shared__ float loss_s[];

  loss_s[tid] = 0.0f;

  float z0_exp, z1_exp, a0, a1;
  int id1, id2;

  for (int i = tid; i < batch_size; i += blockDim.x) {
    id1 = row_major ? i * feature_dim : i;
    id2 = row_major ? i * feature_dim + 1 : i + batch_size;
    z0_exp = exp((double)input[id1]);
    z1_exp = exp((double)input[id2]);

    a0 = z0_exp / (z0_exp + z1_exp);
    a1 = z1_exp / (z0_exp + z1_exp);

    bool no_click = label[i] < 0.5f;

    // calculate the grad

    input[id1] = (a0 - (no_click ? 1.0f : 0.0f)) / batch_size * scaler / total_gpu_count;
    input[id2] = (a1 - (!no_click ? 1.0f : 0.0f)) / batch_size * scaler / total_gpu_count;

    loss_s[tid] += -1 * log(no_click ? a0 : a1);
  }
  __syncthreads();

  float loss_tmp = 0.0f;

  if (tid == 0) {
    for (int i = 0; i < blockDim.x; ++i) loss_tmp += loss_s[i];
    cel_loss[0] = loss_tmp / batch_size + rterm;
  }
}

void CrossEntropyLoss::do_fused_loss_computation(float *input, const float *label, float *loss,
                                                 int batch_size, int feature_dim, float scaler,
                                                 float rterm, hipStream_t stream) {
  bool row_major = (input_tensors_[0]->get_format() == TensorFormat_t::HW);
  int block_size = min(batch_size, 1024);
  CrossEntropy_Kernel<<<1, block_size, block_size * sizeof(float), stream>>>(
      input, label, loss, batch_size, total_gpu_count_, feature_dim, row_major, scaler, rterm);
}

BinaryCrossEntropyLoss::BinaryCrossEntropyLoss(
    const std::shared_ptr<const Tensor<float>> &label_tensor,
    const std::shared_ptr<Tensor<float>> &input_tensor,
    const std::shared_ptr<Tensor<float>> &loss_tensor,
    const std::shared_ptr<Regularizer> regularizer, int device_id, int total_gpu_count,
    float scaler)
    : Loss(label_tensor, input_tensor, loss_tensor, regularizer, device_id, total_gpu_count,
           scaler) {
  if (input_tensor->get_format() != label_tensor->get_format())
    CK_THROW_(Error_t::WrongInput, "Format of input tensor and label tensor don't match");

  bool row_major = (input_tensor->get_format() == TensorFormat_t::HW);
  const auto &input_dim = input_tensor->get_dims();
  int feature_dim = row_major ? input_dim[1] : input_dim[0];
  if (feature_dim != 1)
    CK_THROW_(Error_t::WrongInput, "The feature dimension of BCE loss input should be 1");
}

// Suppose we use one thread to calculate one sample
__global__ void BinaryCrossEntropy_Kernel(float *input, const float *label, float *bce_loss,
                                          float scaler, int batch_size, int total_gpu_count,
                                          float rterm) {
  int tid = threadIdx.x;
  extern __shared__ float loss_s[];
  loss_s[tid] = 0.0f;

  for (int i = tid; i < batch_size; i += blockDim.x) {
    const float x = input[i];

    const float y = label[i];
    if (x >= 0) {
      float exp_neg_x = exp(-x);
      loss_s[tid] += x * (1 - y) + log(1 + exp_neg_x);
      input[i] = ((1.f - y) - exp_neg_x / (1.f + exp_neg_x)) * scaler / (float)batch_size /
                 total_gpu_count;
    } else {
      float exp_x = exp(x);
      loss_s[tid] += -x * y + log(1 + exp_x);
      input[i] = (-y + exp_x / (1.f + exp_x)) * scaler / (float)batch_size / total_gpu_count;
    }
  }
  __syncthreads();

  float loss_tmp = 0.0f;
  if (tid == 0) {
    for (int i = 0; i < blockDim.x; ++i) loss_tmp += loss_s[i];
    bce_loss[0] = loss_tmp / batch_size + rterm;
  }
}

void BinaryCrossEntropyLoss::do_fused_loss_computation(float *input, const float *label,
                                                       float *loss, int batch_size, int feature_dim,
                                                       float scaler, float rterm,
                                                       hipStream_t stream) {
  int block_size = min(batch_size, 1024);
  BinaryCrossEntropy_Kernel<<<1, block_size, block_size * sizeof(float), stream>>>(
      input, label, loss, scaler, batch_size, total_gpu_count_, rterm);
}

__forceinline__ __device__ __host__ float cross_entropy_loss(float x, float y) {
  float loss = 0.f;
  if (x >= 0) {
    float exp_neg_x = exp(-x);
    loss = x * (1 - y) + log(1 + exp_neg_x);
  } else {
    float exp_x = exp(x);
    loss = -x * y + log(1 + exp_x);
  }
  return -loss;
}

__forceinline__ __device__ __host__ float cross_entropy_loss_backward(float x, float y) {
  float grad = 0.f;
  if (x >= 0) {
    float exp_neg_x = exp(-x);
    grad = ((1 - y) - exp_neg_x / (1 + exp_neg_x));
  } else {
    float exp_x = exp(x);
    grad = (-y + exp_x / (1 + exp_x));
  }
  return grad;
}

__global__ void MultiCrossEntropy_Kernel(float *input, const float *label,
                                         const float *target_weight, float *bce_loss, int batchsize,
                                         int total_gpu_count, int labels_per_sample, float scaler,
                                         float rterm) {
  int tid = threadIdx.x + blockDim.x * blockIdx.x;
  int num_threads = blockDim.x * gridDim.x;
  float loss_s = 0.f;
  const int size = batchsize * labels_per_sample;
  for (int i = tid; i < size; i += num_threads) {
    int target_weight_idx = i % labels_per_sample;
    const float x = input[i];
    const float y = label[i];
    float loss =
        (label[i] < -0.5) ? 0.f : (target_weight[target_weight_idx] * cross_entropy_loss(x, y));
    loss_s += loss;
    input[i] = (label[i] < -0.5)
                   ? 0.f
                   : (target_weight[target_weight_idx] * cross_entropy_loss_backward(x, y) / size *
                      scaler / total_gpu_count);
  }

  atomic_global_sum_div(-loss_s, bce_loss, size);
  if (tid == 0) {
    atomicAdd(bce_loss, rterm);
  }
  return;
}

void MultiCrossEntropyLoss::do_fused_loss_computation(float *input, const float *label, float *loss,
                                                      int batch_size, int feature_dim, float scaler,
                                                      float rterm, hipStream_t stream) {
  int labels_per_sample = feature_dim;
  hipMemsetAsync(loss, 0, loss_tensors_[0]->get_size(), stream);

  const int BLOCK_SIZE = 256;
  const int GRID_SIZE = min(40, (batch_size * labels_per_sample - 1) / BLOCK_SIZE);
  float *target_weight = target_weight_->get_ptr();
  MultiCrossEntropy_Kernel<<<GRID_SIZE, BLOCK_SIZE, 0, stream>>>(input, label, target_weight, loss,
                                                                 batch_size, total_gpu_count_,
                                                                 labels_per_sample, scaler, rterm);
}

MultiCrossEntropyLoss::MultiCrossEntropyLoss(
    const std::shared_ptr<const Tensor<float>> &label_tensor,
    const std::shared_ptr<Tensor<float>> &input_tensor,
    const std::shared_ptr<Tensor<float>> &loss_tensor,
    const std::shared_ptr<Regularizer> regularizer, const std::vector<float> &target_weight,
    int device_id, int total_gpu_count, float scaler)
    : Loss(label_tensor, input_tensor, loss_tensor, regularizer, device_id, total_gpu_count,
           scaler) {
  if (label_tensor->get_dims().size() != 2 || label_tensor->get_format() != TensorFormat_t::HW ||
      input_tensor->get_dims().size() != 2 || input_tensor->get_format() != TensorFormat_t::HW ||
      label_tensor->get_dims()[0] != input_tensor->get_dims()[0] ||
      label_tensor->get_dims()[1] != input_tensor->get_dims()[1]) {
    CK_THROW_(Error_t::WrongInput, "Format of input tensor and label tensor don't match");
  }
  // verify the length of target_weight
  if (target_weight.size() != input_tensor->get_dims()[1]) {
    CK_THROW_(Error_t::WrongInput, "target_weight.size() != input_tensor.get_dims()[0]");
  }

  // load target_weight to internal Tensor
  internal_buff_.reset(new GeneralBuffer<float>());
  std::vector<size_t> twdim = {1, label_tensor->get_dims()[1]};
  target_weight_.reset(new Tensor<float>(twdim, internal_buff_, TensorFormat_t::HW));
  internal_buff_->init(device_id);
  CudaDeviceContext context(device_id);
  CK_CUDA_THROW_(hipMemcpy(target_weight_->get_ptr(), target_weight.data(),
                            target_weight_->get_size(), hipMemcpyHostToDevice));

  return;
}
}  // namespace HugeCTR
