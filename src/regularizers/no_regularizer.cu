/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "HugeCTR/include/regularizers/no_regularizer.hpp"

#include "HugeCTR/include/utils.cuh"

#include <utility>

#ifndef NDEBUG
#include <iostream>
#endif

namespace HugeCTR {

NoRegularizer::NoRegularizer(const std::shared_ptr<GeneralBuffer<float>>& weight_buff,
                             const std::shared_ptr<GeneralBuffer<float>>& wgrad_buff,
                             const int batch_size, const int device_id)
    : Regularizer(weight_buff, wgrad_buff, batch_size, device_id) {}

void NoRegularizer::do_compute_rterm(const float* weight, float* rterm, int num_elements,
                                     hipStream_t stream) {
  *rterm = 0.0f;
}

void NoRegularizer::do_initialize_wgrad(const float* weight, float* wgrad, int num_elements,
                                        hipStream_t stream) {
  int n_blocks = get_n_sms() * 4;
  int block_size = 512;
  initialize_array<<<n_blocks, block_size, 0, stream>>>(wgrad, num_elements, 0.0f);
}

}  // namespace HugeCTR
