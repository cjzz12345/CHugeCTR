#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "HugeCTR/include/optimizers/nesterov_optimizer.hpp"

namespace {

__global__ void nesterov_kernel(int len, float* weight, const float* wgrad, float* accum, float lr,
                                float mu, float scaler) {
  const int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < len) {
    float accum_old = accum[i];
    float accum_new = mu * accum_old - lr * wgrad[i] / scaler;
    accum[i] = accum_new;
    weight[i] += (-mu * accum_old + (1 + mu) * accum_new);
  }
}

}  // namespace

namespace HugeCTR {

void NesterovOptimizer::update(hipStream_t stream) {
  CudaDeviceContext context(device_id_);

  const int len = weight_->get_num_elements();
  const int block_dim = 256;
  const int grid_dim = (len - 1) / block_dim + 1;

  float* weight = weight_->get_ptr_with_offset(0);
  const float* wgrad = wgrad_->get_ptr_with_offset(0);
  float* accum = accum_.get_ptr_with_offset(0);

  nesterov_kernel<<<grid_dim, block_dim, 0, stream>>>(len, weight, wgrad, accum, lr_, mu_, scaler_);

#ifndef NDEBUG
  hipDeviceSynchronize();
  CK_CUDA_THROW_(hipGetLastError());
#endif
}

}  // namespace HugeCTR
