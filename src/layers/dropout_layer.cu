#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "HugeCTR/include/layers/dropout_layer.hpp"

#include <algorithm>
#include <cstdio>
#include <ctime>
#include <functional>
#include "HugeCTR/include/utils.cuh"
#include "HugeCTR/include/utils.hpp"
#ifndef NDEBUG
#include <iostream>
#endif

namespace HugeCTR {

namespace {

__global__ void dropout_kernel(const float* in, const float* mask, float* out, const int len,
                               const float rate, const float scale) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < len; i += blockDim.x * gridDim.x) {
    out[i] = ((1.f - mask[i]) >= rate) * in[i] * scale;
  }
}

}  // end namespace

DropoutLayer::DropoutLayer(const std::shared_ptr<Tensor<float>>& in_tensor,
                           const std::shared_ptr<Tensor<float>>& out_tensor, float rate,
                           const hiprandGenerator_t& curand_generator, int device_id)
    : Layer(device_id),
      rate_(rate),
      scale_(1.0 / (1.0 - rate)),
      mask_(nullptr),
      curand_generator_(curand_generator),
      n_sms_(0) {
  assert(get_size_from_dims(in_tensor->get_dims()) == get_size_from_dims(out_tensor->get_dims()));
  assert(rate_ > 0.f && rate_ < 1.f);

  in_tensors_.emplace_back(in_tensor);
  out_tensors_.emplace_back(out_tensor);

  CudaDeviceContext context(get_device_id());
  CK_CUDA_THROW_(hipMalloc(&mask_, in_tensor->get_size()));
  CK_CURAND_THROW_(hiprandSetPseudoRandomGeneratorSeed(curand_generator_, get_seed()));

  CK_CUDA_THROW_(hipDeviceGetAttribute(&n_sms_, hipDeviceAttributeMultiprocessorCount, get_device_id()));
  assert(n_sms_ > 0);
}

DropoutLayer::~DropoutLayer() {
  if (mask_) {
    hipFree(mask_);
  }
}

void DropoutLayer::fprop(hipStream_t stream) {
  CudaDeviceContext context(get_device_id());
  CK_CURAND_THROW_(
      hiprandGenerateUniform(curand_generator_, mask_, in_tensors_[0]->get_num_elements()));
  prop_common(in_tensors_[0]->get_ptr(), out_tensors_[0]->get_ptr(), stream);
}

void DropoutLayer::bprop(hipStream_t stream) {
  CudaDeviceContext context(get_device_id());
  prop_common(out_tensors_[0]->get_ptr(), in_tensors_[0]->get_ptr(), stream);
}

void DropoutLayer::inference(hipStream_t stream) {
  CudaDeviceContext context(get_device_id());
  hipMemcpyAsync(out_tensors_[0]->get_ptr(), in_tensors_[0]->get_ptr(), in_tensors_[0]->get_size(),
                  hipMemcpyDeviceToDevice, stream);
}

int64_t DropoutLayer::get_seed() const {
  FILE* f = fopen("/dev/urandom", "rb");
  if (f) {
    int64_t seed;
    size_t ret = fread(&seed, 1, sizeof(seed), f);
    fclose(f);
    if (ret == sizeof(seed)) {
      return seed;
    }
  }
  return time(nullptr);
}

void DropoutLayer::prop_common(const float* in, float* out, hipStream_t stream) {
  int len = in_tensors_[0]->get_num_elements();

  int grid_size = n_sms_ * 16;
  int block_size = 256;
  dropout_kernel<<<grid_size, block_size, 0, stream>>>(in, mask_, out, len, rate_, scale_);

#ifndef NDEBUG
  hipDeviceSynchronize();
  CK_CUDA_THROW_(hipGetLastError());
#endif
}

}  // namespace HugeCTR
