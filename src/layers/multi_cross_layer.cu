#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <math.h>
#include <vector>
#include "HugeCTR/include/layers/multi_cross_layer.hpp"
#include "HugeCTR/include/utils.cuh"

namespace HugeCTR {

// kernels
namespace {

/**
 * Each row in `mat`dot product with vec, length of vec should be w. Then adding bias for each of
 * the rows
 * @param out: hx1
 * @param mat: hxw
 * @param vec: 1xw
 */
__global__ void matrix_vec_mul_kernel(float* out, const float* mat, int h, int w,
                                      const float* vec) {
  const int tid = blockDim.x * blockIdx.x + threadIdx.x;
  const int wtid = tid % WARP_SIZE;  // thread id in warp
  const int wid = tid / WARP_SIZE;   // warp id
  const float* mat_with_offset = mat + wid * w;
  if (wid < h) {
    float accum = 0.;
    for (int i = wtid; i < w; i += WARP_SIZE) {
      accum += mat_with_offset[i] * vec[i];
    }
    float val = warpReduceSum(accum);
    if (wtid == 0) {
      out[wid] = val;
    }
  }
}

void matrix_vec_mul(Tensor<float>& out, const Tensor<float>& mat, const Tensor<float>& vec,
                    hipStream_t stream) {
  float* pout = out.get_ptr();
  const float* pmat = mat.get_ptr();
  const float* pvec = vec.get_ptr();

  const auto& dim = out.get_dims();
  const auto& idim = mat.get_dims();
  assert(dim.size() == 2 && idim.size() == 2 && idim[1] == vec.get_dims()[1] &&
         vec.get_dims()[0] == 1);
  assert(idim[0] == dim[0]);

  const int h = idim[0];
  const int w = idim[1];

  const int BLOCK_DIM = 256;
  const int GRID_DIM = calc_grid(h * WARP_SIZE, BLOCK_DIM);

  matrix_vec_mul_kernel<<<GRID_DIM, BLOCK_DIM, 0, stream>>>(pout, pmat, h, w, pvec);
}

/**
 * Each row in `mat` scale with the coresponding element in vec.
 * The length of vec should be h.
 * @param o_mat: hxw
 * @param mat: hxw
 * @param vec: hx1
 */
__global__ void row_scaling_kenrel(float* o_mat, const float* mat, int h, int w, const float* vec) {
  const int tid = blockDim.x * blockIdx.x + threadIdx.x;
  if (tid < h * w) {
    const int row = tid / w;
    o_mat[tid] = mat[tid] * vec[row];
  }
}

void row_scaling(Tensor<float>& o_mat, const Tensor<float>& mat, const Tensor<float>& vec,
                 hipStream_t stream) {
  float* pout = o_mat.get_ptr();
  const float* pmat = mat.get_ptr();
  const float* pvec = vec.get_ptr();

  const auto& dim = o_mat.get_dims();
  const auto& idim = mat.get_dims();
  assert(dim.size() == 2 && idim.size() == 2 && dim[0] == vec.get_dims()[0] &&
         vec.get_dims()[1] == 1);
  assert(idim[0] == dim[0] && idim[1] == dim[1]);

  const int h = dim[0];
  const int w = dim[1];

  const int BLOCK_DIM = 256;
  const int GRID_DIM = calc_grid(h * w, BLOCK_DIM);

  row_scaling_kenrel<<<GRID_DIM, BLOCK_DIM, 0, stream>>>(pout, pmat, h, w, pvec);
}

/**
 * Each row in `mat` sum with  vec.
 * The length of vec should be w.
 * @param o_mat: hxw
 * @param mat: hxw
 * @param vec: 1xw
 */
__global__ void matrix_vec_add_kenrel(float* o_mat, const float* mat, int h, int w,
                                      const float* vec) {
  const int tid = blockDim.x * blockIdx.x + threadIdx.x;
  if (tid < h * w) {
    const int col = tid % w;
    o_mat[tid] = mat[tid] + vec[col];
  }
}

void matrix_vec_add(Tensor<float>& o_mat, const Tensor<float>& mat, const Tensor<float>& vec,
                    hipStream_t stream) {
  float* pout = o_mat.get_ptr();
  const float* pmat = mat.get_ptr();
  const float* pvec = vec.get_ptr();

  const auto& dim = o_mat.get_dims();
  const auto& idim = mat.get_dims();
  assert(dim.size() == 2 && idim.size() == 2 && dim[1] == vec.get_dims()[1] &&
         vec.get_dims()[0] == 1);
  assert(idim[0] == dim[0] && idim[1] == dim[1]);

  const int h = dim[0];
  const int w = dim[1];

  const int BLOCK_DIM = 256;
  const int GRID_DIM = calc_grid(h * w, BLOCK_DIM);

  matrix_vec_add_kenrel<<<GRID_DIM, BLOCK_DIM, 0, stream>>>(pout, pmat, h, w, pvec);
}

/**
 * Pointwise adding
 */
__global__ void matrix_add_kenrel(float* o_mat, const float* mat_a, int h, int w,
                                  const float* mat_b) {
  const int tid = blockDim.x * blockIdx.x + threadIdx.x;
  if (tid < h * w) {
    o_mat[tid] = mat_a[tid] + mat_b[tid];
  }
}

void matrix_add(Tensor<float>& out_mat, const Tensor<float>& mat_a, const Tensor<float>& mat_b,
                hipStream_t stream) {
  float* pout = out_mat.get_ptr();
  const float* pmat_a = mat_a.get_ptr();
  const float* pmat_b = mat_b.get_ptr();

  const auto& dim = out_mat.get_dims();
  const auto& idim1 = mat_a.get_dims();
  const auto& idim2 = mat_b.get_dims();
  assert(idim1[0] == dim[0] && idim1[1] == dim[1]);
  assert(idim2[0] == dim[0] && idim2[1] == dim[1]);

  const int h = dim[0];
  const int w = dim[1];

  const int BLOCK_DIM = 256;
  const int GRID_DIM = calc_grid(h * w, BLOCK_DIM);
  matrix_add_kenrel<<<GRID_DIM, BLOCK_DIM, 0, stream>>>(pout, pmat_a, h, w, pmat_b);
}

/**
 * compute dot product for each pair of the rows in the two matrix,
 */
__global__ void matrix_pair_mul_kernel(float* o_vec, const float* mat_a, int h, int w,
                                       const float* mat_b) {
  const int tid = blockDim.x * blockIdx.x + threadIdx.x;
  const int wtid = tid % WARP_SIZE;  // thread id in warp
  const int wid = tid / WARP_SIZE;   // warp id
  const float* mat_a_with_offset = mat_a + wid * w;
  const float* mat_b_with_offset = mat_b + wid * w;
  if (wid < h) {
    float accum = 0.f;
    for (int i = wtid; i < w; i += WARP_SIZE) {
      accum += mat_a_with_offset[i] * mat_b_with_offset[i];
    }
    float val = warpReduceSum(accum);
    if (wtid == 0) {
      o_vec[wid] = val;
    }
  }
}

void matrix_pair_mul(Tensor<float>& o_vec, const Tensor<float>& mat_a, const Tensor<float>& mat_b,
                     hipStream_t stream) {
  float* pout = o_vec.get_ptr();
  const float* pmat_a = mat_a.get_ptr();
  const float* pmat_b = mat_b.get_ptr();

  const auto& dim = mat_a.get_dims();

  const int h = dim[0];
  const int w = dim[1];
  assert(h == mat_b.get_dims()[0] && w == mat_a.get_dims()[1] && h == o_vec.get_dims()[0] &&
         1 == o_vec.get_dims()[1]);

  const int BLOCK_DIM = 256;
  const int GRID_DIM = calc_grid(h * WARP_SIZE, BLOCK_DIM);
  matrix_pair_mul_kernel<<<GRID_DIM, BLOCK_DIM, 0, stream>>>(pout, pmat_a, h, w, pmat_b);
}

/**
 * out product of two vectors
 * @param out_mat: hxw
 * @param vec_a: hx1
 * @param vec_b: 1xw
 */
__global__ void out_product_kernel(float* out_mat, const float* vec_a, int h, const float* vec_b,
                                   int w) {
  const int tid = blockDim.x * blockIdx.x + threadIdx.x;
  if (tid < h * w) {
    const int col = tid % w;
    const int row = tid / w;
    out_mat[tid] = vec_a[row] * vec_b[col];
  }
}
void out_product(Tensor<float>& out_mat, const Tensor<float>& vec_a, const Tensor<float>& vec_b,
                 hipStream_t stream) {
  float* pout = out_mat.get_ptr();
  const float* pvec_a = vec_a.get_ptr();
  const float* pvec_b = vec_b.get_ptr();
  const auto& dim = out_mat.get_dims();

  const int h = dim[0];
  const int w = dim[1];

  assert(h == vec_a.get_dims()[0] && w == vec_b.get_dims()[1] && vec_a.get_dims()[1] == 1 &&
         vec_b.get_dims()[0] == 1);
  const int BLOCK_DIM = 256;
  const int GRID_DIM = calc_grid(h * w, BLOCK_DIM);
  out_product_kernel<<<GRID_DIM, BLOCK_DIM, 0, stream>>>(pout, pvec_a, h, pvec_b, w);
}

/**
 * Each row in `mat` scale with the coresponding element in vec. and accum across rows
 * The length of vec should be h.
 * @param o_mat: hxw
 * @param mat: hxw
 * @param vec: hx1
 */
__global__ void row_scaling_sum_kernel(float* out, const float* mat, int h, int w,
                                       const float* vec) {
  const int tid = blockDim.x * blockIdx.x + threadIdx.x;
  const int wtid = tid % WARP_SIZE;  // thread id in warp
  const int wid = tid / WARP_SIZE;   // warp id
  if (wid < w) {
    float accum = 0.f;
    for (int i = wtid; i < h; i += WARP_SIZE) {
      const int col = wid;
      const int idx = i * w + col;
      accum += mat[idx] * vec[i];
    }
    float val = warpReduceSum(accum);
    if (wtid == 0) {
      out[wid] += val;  // using += here to enable regularization
    }
  }
}

void row_scaling_sum(Tensor<float>& out, const Tensor<float>& mat, const Tensor<float>& vec,
                     hipStream_t stream) {
  float* pout = out.get_ptr();
  const float* pmat = mat.get_ptr();
  const float* pvec = vec.get_ptr();

  const auto& dim = out.get_dims();
  const auto& idim = mat.get_dims();
  assert(dim.size() == 2 && idim.size() == 2 && idim[0] == vec.get_dims()[0] &&
         vec.get_dims()[1] == 1);
  assert(idim[1] == dim[1]);

  const int h = idim[0];
  const int w = idim[1];

  const int BLOCK_DIM = 256;
  const int GRID_DIM = calc_grid(w * WARP_SIZE, BLOCK_DIM);  // each col one warp

  row_scaling_sum_kernel<<<GRID_DIM, BLOCK_DIM, 0, stream>>>(pout, pmat, h, w, pvec);
}

/**
 * Accum across rows
 * @param o_mat: 1xw
 * @param mat: hxw
 */
__global__ void row_sum_kernel(float* out, const float* mat, int h, int w) {
  const int tid = blockDim.x * blockIdx.x + threadIdx.x;
  const int wtid = tid % WARP_SIZE;  // thread id in warp
  const int wid = tid / WARP_SIZE;   // warp id
  if (wid < w) {
    float accum = 0.f;
    for (int i = wtid; i < h; i += WARP_SIZE) {
      const int col = wid;
      const int idx = i * w + col;
      accum += mat[idx];
    }
    float val = warpReduceSum(accum);
    if (wtid == 0) {
      out[wid] += val;  // using += here to enable regularization
    }
  }
}

void rows_sum(Tensor<float>& out, const Tensor<float>& mat, hipStream_t stream) {
  float* pout = out.get_ptr();
  const float* pmat = mat.get_ptr();

  const auto& dim = out.get_dims();
  const auto& idim = mat.get_dims();
  assert(dim.size() == 2 && idim.size() == 2);
  assert(idim[1] == dim[1]);

  const int h = idim[0];
  const int w = idim[1];

  const int BLOCK_DIM = 256;
  const int GRID_DIM = calc_grid(w * WARP_SIZE, BLOCK_DIM);  // each col one warp

  row_sum_kernel<<<GRID_DIM, BLOCK_DIM, 0, stream>>>(pout, pmat, h, w);
}

}  // namespace

/*
 * Equivalent TensorFlow Code:
 *
def forward(x, k, b, layers):
  y = []
  h = []
  for i in range(layers):
    v = tf.linalg.matvec(x if i == 0 else y[i - 1], k[i])
    v = tf.transpose(v)
    h.append(v)
    m = tf.multiply(x, v)
    m = tf.add(m, x if i == 0 else y[i - 1])
    m = tf.add(m, b[i])
    y.append(m)
  return y, h
 *
 */
void MultiCrossForwardFunctor::operator()(hipStream_t stream, const Tensor<float>& input_tensor,
                                          const std::vector<const Tensor<float>*>& kernel_tensors,
                                          const std::vector<const Tensor<float>*>& bias_tensors,
                                          const std::vector<Tensor<float>*>& layer_output_tensors,
                                          const std::vector<Tensor<float>*>& layer_hidden_tensors,
                                          int num_layers) const {
  for (int i = 0; i < num_layers; i++) {
    matrix_vec_mul(*layer_hidden_tensors[i], i == 0 ? input_tensor : *layer_output_tensors[i - 1],
                   *kernel_tensors[i], stream);
    row_scaling(*layer_output_tensors[i], input_tensor, *layer_hidden_tensors[i], stream);
    matrix_add(*layer_output_tensors[i], *layer_output_tensors[i],
               i == 0 ? input_tensor : *layer_output_tensors[i - 1], stream);
    matrix_vec_add(*layer_output_tensors[i], *layer_output_tensors[i], *bias_tensors[i], stream);
  }
}

/*
 * Equivalent TensorFlow Code:
 *
def backward(x, k, y, h, dy, layers):
  dx = tf.zeros(x.shape)
  dk = []
  db = []
  for i in reversed(range(layers)):
    dx = tf.add(dx, tf.multiply(dy, h[i]))
    dv = tf.expand_dims(tf.reduce_sum(tf.multiply(dy, x), 1), 1)
    dk.insert(0, tf.linalg.matvec(x if i == 0 else y[i - 1], tf.transpose(dv), transpose_a=True))
    db.insert(0, tf.expand_dims(tf.reduce_sum(dy, 0), 0))
    dy = tf.add(dy, tf.matmul(dv, k[i]))
  dx = tf.add(dx, dy)
  return dx, dk, db
 *
 */
void MultiCrossBackwardFunctor::operator()(
    hipStream_t stream, const Tensor<float>& input_tensor,
    const std::vector<const Tensor<float>*>& kernel_tensors,
    const std::vector<const Tensor<float>*>& layer_output_tensors,
    const std::vector<const Tensor<float>*>& layer_hidden_tensors, const Tensor<float>& grad_tensor,
    Tensor<float>& output_tensor, const std::vector<Tensor<float>*>& kernel_output_tensors,
    const std::vector<Tensor<float>*>& bias_output_tensors, Tensor<float>& tmp_vec_tensor,
    const std::vector<Tensor<float>*>& tmp_mat_tensors, int num_layers) const {
  hipMemsetAsync(tmp_mat_tensors[2]->get_ptr(), 0, tmp_mat_tensors[2]->get_size(), stream);
  for (int i = num_layers - 1; i >= 0; i--) {
    row_scaling(*tmp_mat_tensors[0], i == num_layers - 1 ? grad_tensor : *tmp_mat_tensors[1],
                *layer_hidden_tensors[i], stream);
    matrix_add(*tmp_mat_tensors[2], *tmp_mat_tensors[2], *tmp_mat_tensors[0], stream);
    matrix_pair_mul(tmp_vec_tensor, i == num_layers - 1 ? grad_tensor : *tmp_mat_tensors[1],
                    input_tensor, stream);
    row_scaling_sum(*kernel_output_tensors[i], i == 0 ? input_tensor : *layer_output_tensors[i - 1],
                    tmp_vec_tensor, stream);
    rows_sum(*bias_output_tensors[i], i == num_layers - 1 ? grad_tensor : *tmp_mat_tensors[1],
             stream);
    out_product(*tmp_mat_tensors[0], tmp_vec_tensor, *kernel_tensors[i], stream);
    matrix_add(*tmp_mat_tensors[1], i == num_layers - 1 ? grad_tensor : *tmp_mat_tensors[1],
               *tmp_mat_tensors[0], stream);
  }
  matrix_add(output_tensor, *tmp_mat_tensors[2], *tmp_mat_tensors[1], stream);
}

MultiCrossLayer::MultiCrossLayer(const GeneralBufferPtr<float>& weight_buff,
                                 const GeneralBufferPtr<float>& wgrad_buff,
                                 const TensorPtr<float>& in_tensor,
                                 const TensorPtr<float>& out_tensor, int num_layers, int device_id)
    : Layer(device_id), num_layers_(num_layers), blobs_buff_(new GeneralBuffer<float>()) {
  try {
    // check the in_tensor and out_tensor
    const auto& in_tensor_dim = in_tensor->get_dims();
    const auto& out_tensor_dim = out_tensor->get_dims();
    // 1. two dim?
    if (in_tensor_dim.size() != 2 || out_tensor_dim.size() != 2) {
      CK_THROW_(Error_t::WrongInput, "input or output tensor doesn't has two dimensions");
    }
    // 2. same dim?
    for (int i = 0; i < 2; i++) {
      if (in_tensor_dim[i] != out_tensor_dim[i]) {
        CK_THROW_(Error_t::WrongInput, "input and output tensor doesn't match");
      }
    }
    size_t vec_length = in_tensor_dim[1];
    size_t batchsize = in_tensor_dim[0];

    // check num_lyaers
    if (num_layers < 1) {
      CK_THROW_(Error_t::WrongInput, "num_layers < 1");
    }

    std::vector<size_t> weight_bias_dim = {1, vec_length};
    for (int i = 0; i < num_layers; i++) {
      // setup weights
      weights_.emplace_back(new Tensor<float>(weight_bias_dim, weight_buff, TensorFormat_t::HW));
      // setup bias
      weights_.emplace_back(new Tensor<float>(weight_bias_dim, weight_buff, TensorFormat_t::HW));
      // setup weight gradient
      wgrad_.emplace_back(new Tensor<float>(weight_bias_dim, wgrad_buff, TensorFormat_t::HW));
      // setup bias gradient
      wgrad_.emplace_back(new Tensor<float>(weight_bias_dim, wgrad_buff, TensorFormat_t::HW));
    }

    in_tensors_.emplace_back(in_tensor);
    out_tensors_.emplace_back(out_tensor);
    // setup blobs
    std::vector<size_t> blob_dim = {batchsize, vec_length};
    blob_tensors_.emplace_back(in_tensor);
    for (int i = 0; i < num_layers - 1; i++) {
      blob_tensors_.emplace_back(new Tensor<float>(blob_dim, blobs_buff_, TensorFormat_t::HW));
    }
    blob_tensors_.emplace_back(out_tensor);

    for (int i = 0; i < 3; i++) {
      tmp_mat_tensors_[i].reset(new Tensor<float>(blob_dim, blobs_buff_, TensorFormat_t::HW));
    }
    std::vector<size_t> tmp_vec_dim = {batchsize, 1};
    tmp_vec_tensor_.reset(new Tensor<float>(tmp_vec_dim, blobs_buff_, TensorFormat_t::HW));
    for (int i = 0; i < num_layers; i++) {
      vec_tensors_.emplace_back(new Tensor<float>(tmp_vec_dim, blobs_buff_, TensorFormat_t::HW));
    }
    blobs_buff_->init(device_id);
  } catch (const std::runtime_error& rt_err) {
    std::cerr << rt_err.what() << std::endl;
    throw;
  }
}

void MultiCrossLayer::fprop(hipStream_t stream) {
  CudaDeviceContext context(get_device_id());
  std::vector<const Tensor<float>*> kernel_tensors;
  std::vector<const Tensor<float>*> bias_tensors;
  std::vector<Tensor<float>*> output_tensors;
  std::vector<Tensor<float>*> hidden_tensors;

  for (int i = 0; i < num_layers_; i++) {
    kernel_tensors.push_back(weights_[2 * i].get());
    bias_tensors.push_back(weights_[2 * i + 1].get());
  }

  for (int i = 0; i < num_layers_; i++) {
    output_tensors.push_back(blob_tensors_[i + 1].get());
    hidden_tensors.push_back(vec_tensors_[i].get());
  }

  MultiCrossForwardFunctor()(stream, *blob_tensors_[0], kernel_tensors, bias_tensors,
                             output_tensors, hidden_tensors, num_layers_);
}

void MultiCrossLayer::bprop(hipStream_t stream) {
  CudaDeviceContext context(get_device_id());
  std::vector<const Tensor<float>*> kernel_tensors;
  std::vector<Tensor<float>*> kernel_output_tensors;
  std::vector<Tensor<float>*> bias_output_tensors;
  std::vector<const Tensor<float>*> forward_output_tensors;
  std::vector<const Tensor<float>*> forward_hidden_tensors;

  for (int i = 0; i < num_layers_; i++) {
    kernel_tensors.push_back(weights_[2 * i].get());
    kernel_output_tensors.push_back(wgrad_[2 * i].get());
    bias_output_tensors.push_back(wgrad_[2 * i + 1].get());
    forward_hidden_tensors.push_back(vec_tensors_[i].get());
  }

  for (int i = 0; i < num_layers_ - 1; i++) {
    forward_output_tensors.push_back(blob_tensors_[i + 1].get());
  }

  MultiCrossBackwardFunctor()(
      stream, *blob_tensors_[0], kernel_tensors, forward_output_tensors, forward_hidden_tensors,
      *blob_tensors_[num_layers_], *blob_tensors_[0], kernel_output_tensors, bias_output_tensors,
      *tmp_vec_tensor_,
      {tmp_mat_tensors_[0].get(), tmp_mat_tensors_[1].get(), tmp_mat_tensors_[2].get()},
      num_layers_);
}

std::vector<float> MultiCrossLayer::get_initializer() {
  std::vector<float> initializer;
  size_t weight_size = 0;
  for (const auto& w : weights_) {
    weight_size += w->get_num_elements();
  }
  initializer.resize(weight_size);
  const auto& in_tensor = in_tensors_[0];
  const auto& out_tensor = out_tensors_[0];
  float in_dim = in_tensor->get_dims()[1];
  float out_dim = out_tensor->get_dims()[1];

  // glorot_uniform
  float limit = sqrt(6.f / (in_dim + out_dim));
  HugeCTR::UnifiedDataSimulator<float> fdata_sim(-1 * limit, limit);

  size_t w_size_accum = 0;
  for (int i = 0; i < num_layers_; i++) {
    // setup weights
    size_t w_size = (weights_[2 * i])->get_num_elements();
    for (unsigned int j = 0; j < w_size; j++) {
      initializer[w_size_accum + j] = fdata_sim.get_num();
    }
    w_size_accum += w_size;
    // setup bias
    w_size = (weights_[2 * i + 1])->get_num_elements();
    for (unsigned int j = 0; j < w_size; j++) {
      initializer[w_size_accum + j] = 0.f;
    }
    w_size_accum += w_size;
  }
  return initializer;
}

}  // namespace HugeCTR
