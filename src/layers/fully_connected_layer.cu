#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "HugeCTR/include/layers/fully_connected_layer.hpp"

#include "HugeCTR/include/utils.cuh"

#include <math.h>
#include <vector>
#include "HugeCTR/include/data_parser.hpp"

namespace HugeCTR {

FullyConnectedLayer::FullyConnectedLayer(const std::shared_ptr<GeneralBuffer<float>>& weight_buff,
                                         const std::shared_ptr<GeneralBuffer<float>>& wgrad_buff,
                                         const std::shared_ptr<Tensor<float>>& in_tensor,
                                         const std::shared_ptr<Tensor<float>>& out_tensor,
                                         TensorFormat_t weight_format,
                                         hipblasHandle_t const& cublas_handle, int device_id,
                                         bool use_mixed_precision)
    : cublas_handle_(cublas_handle), Layer(device_id), use_mixed_precision_(use_mixed_precision) {
  try {
    // check the in_tensor and out_tensor
    const auto& in_tensor_dim = in_tensor->get_dims();
    const auto& out_tensor_dim = out_tensor->get_dims();
    // 1. two dim?
    if (in_tensor_dim.size() != 2 || out_tensor_dim.size() != 2) {
      CK_THROW_(Error_t::WrongInput, "input or output tensor doesn't has two dimensions");
    }
    // 2. dim match?
    assert(in_tensor->get_format() == TensorFormat_t::WH ||
           in_tensor->get_format() == TensorFormat_t::HW);
    assert(out_tensor->get_format() == TensorFormat_t::WH ||
           out_tensor->get_format() == TensorFormat_t::HW);
    size_t m = in_tensor->get_format() == TensorFormat_t::WH ? in_tensor_dim[1] : in_tensor_dim[0];
    size_t n =
        out_tensor->get_format() == TensorFormat_t::WH ? out_tensor_dim[0] : out_tensor_dim[1];
    size_t k = in_tensor->get_format() == TensorFormat_t::WH ? in_tensor_dim[0] : in_tensor_dim[1];
    size_t m_ck =
        out_tensor->get_format() == TensorFormat_t::WH ? out_tensor_dim[1] : out_tensor_dim[0];
    if (m != m_ck) {
      CK_THROW_(Error_t::WrongInput, "size of input / output tensor doesn't match");
    }

    std::vector<size_t> weight_dim;
    std::vector<size_t> bias_dim;
    if (weight_format == TensorFormat_t::WH) {
      weight_dim = {n, k};
      bias_dim = {n, 1};
    } else if (weight_format == TensorFormat_t::HW) {
      weight_dim = {k, n};
      bias_dim = {1, n};
    } else {
      CK_THROW_(Error_t::WrongInput, "weight_format doesn't match Mlp Layer");
    }

    weights_.emplace_back(new Tensor<float>(weight_dim, weight_buff, weight_format));
    weights_.emplace_back(new Tensor<float>(bias_dim, weight_buff, weight_format));
    wgrad_.emplace_back(new Tensor<float>(weight_dim, wgrad_buff, weight_format));
    wgrad_.emplace_back(new Tensor<float>(bias_dim, wgrad_buff, weight_format));
    in_tensors_.emplace_back(in_tensor);
    out_tensors_.emplace_back(out_tensor);
    // Where should we create this cuBLAS handle?
  } catch (const std::runtime_error& rt_err) {
    std::cerr << rt_err.what() << std::endl;
    throw;
  }
}
void __global__ add_bias_kernel_row(float* data, const float* bias, const int m, const int n) {
  int offset = blockIdx.x * n;
  for (int tid = threadIdx.x; tid < n; tid += blockDim.x) {
    data[offset + tid] += bias[tid];
  }
}
void __global__ add_bias_kernel_col(float* data, const float* bias, const int m, const int n) {
  int offset = blockIdx.x * m;
  float b = bias[blockIdx.x];
  for (int tid = threadIdx.x; tid < m; tid += blockDim.x) {
    data[offset + tid] += b;
  }
}
void add_bias(float* data, const float* bias, const int m, const int n, bool row_major,
              hipStream_t stream) {
  if (row_major) {
    dim3 grid(m);
    dim3 block(min(n, 1024));
    add_bias_kernel_row<<<grid, block, 0, stream>>>(data, bias, m, n);
  } else {
    dim3 grid(n);
    dim3 block(min(m, 1024));
    add_bias_kernel_col<<<grid, block, 0, stream>>>(data, bias, m, n);
  }
#ifndef NDEBUG
  hipDeviceSynchronize();
  CK_CUDA_THROW_(hipGetLastError());
#endif
}

void FullyConnectedLayer::fprop(hipStream_t stream) {
  CK_CUBLAS_THROW_(hipblasSetStream(cublas_handle_, stream));
  CudaDeviceContext context(get_device_id());

  const auto& in_tensor = in_tensors_[0];
  const auto& out_tensor = out_tensors_[0];

  float* weight = (weights_[0])->get_ptr();
  float* bias = (weights_[1])->get_ptr();
  float* in = in_tensor->get_ptr();
  float* out = out_tensor->get_ptr();

  const auto& in_tensor_dim = in_tensor->get_dims();
  const auto& out_tensor_dim = out_tensor->get_dims();

  int m, n, k;

  m = in_tensor->get_format() == TensorFormat_t::WH ? in_tensor_dim[1] : in_tensor_dim[0];
  n = out_tensor->get_format() == TensorFormat_t::WH ? out_tensor_dim[0] : out_tensor_dim[1];
  k = in_tensor->get_format() == TensorFormat_t::WH ? in_tensor_dim[0] : in_tensor_dim[1];

  float alpha = 1.0f, beta = 0.0f;

  hipblasGemmAlgo_t algo;
  if (use_mixed_precision_) {
    algo = CUBLAS_GEMM_DEFAULT_TENSOR_OP;
  } else {
    algo = HIPBLAS_GEMM_DEFAULT;
  }

  if ((weights_[0])->get_format() == TensorFormat_t::HW &&
      in_tensor->get_format() == TensorFormat_t::HW &&
      out_tensor->get_format() == TensorFormat_t::HW) {
    CK_CUBLAS_THROW_(hipblasGemmEx(cublas_handle_, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k, &alpha, weight,
                                  HIP_R_32F, n, in, HIP_R_32F, k, &beta, out, HIP_R_32F, n,
                                  HIP_R_32F, algo));
    add_bias(out, bias, m, n, true, stream);
  } else if ((weights_[0])->get_format() == TensorFormat_t::WH &&
             in_tensor->get_format() == TensorFormat_t::WH &&
             out_tensor->get_format() == TensorFormat_t::WH) {
    CK_CUBLAS_THROW_(hipblasGemmEx(cublas_handle_, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, in,
                                  HIP_R_32F, m, weight, HIP_R_32F, k, &beta, out, HIP_R_32F, m,
                                  HIP_R_32F, algo));
    add_bias(out, bias, m, n, false, stream);
  } else
    CK_THROW_(Error_t::UnSupportedFormat, "The format combination is not supported");
}

void __global__ cal_bias_grad_kernel_col(float* out, float* bias_grad, int m, int n,
                                         bool row_major) {
  float local_sum = 0.0f;
  if (!row_major) {
    int offset = blockIdx.x * m;
    for (int tid = threadIdx.x; tid < m; tid += blockDim.x) local_sum += out[tid + offset];
  } else {
    for (int tid = threadIdx.x; tid < m; tid += blockDim.x) local_sum += out[tid * n + blockIdx.x];
  }
  __syncthreads();
  local_sum = blockReduceSum(local_sum);
  if (threadIdx.x == 0) {
    bias_grad[blockIdx.x] += local_sum;
  }
}
void cal_bias_grad(float* out, float* bias_grad, int m, int n, bool row_major,
                   hipStream_t stream) {
  dim3 grid(n);
  dim3 block(1024);
  cal_bias_grad_kernel_col<<<grid, block, 0, stream>>>(out, bias_grad, m, n, row_major);
#ifndef NDEBUG
  hipDeviceSynchronize();
  CK_CUDA_THROW_(hipGetLastError());
#endif
}

void FullyConnectedLayer::bprop(hipStream_t stream) {
  CK_CUBLAS_THROW_(hipblasSetStream(cublas_handle_, stream));

  CudaDeviceContext context(get_device_id());

  const auto& in_tensor = in_tensors_[0];
  const auto& out_tensor = out_tensors_[0];

  float* wgrad = (wgrad_[0])->get_ptr();
  float* bias_grad = (wgrad_[1])->get_ptr();
  float* weight = (weights_[0])->get_ptr();
  float* in = in_tensor->get_ptr();
  float* out = out_tensor->get_ptr();

  const auto& in_tensor_dim = in_tensor->get_dims();
  const auto& out_tensor_dim = out_tensor->get_dims();

  int m, n, k;

  m = in_tensor->get_format() == TensorFormat_t::WH ? in_tensor_dim[1] : in_tensor_dim[0];
  n = out_tensor->get_format() == TensorFormat_t::WH ? out_tensor_dim[0] : out_tensor_dim[1];
  k = in_tensor->get_format() == TensorFormat_t::WH ? in_tensor_dim[0] : in_tensor_dim[1];

  hipblasGemmAlgo_t algo;
  if (use_mixed_precision_) {
    algo = CUBLAS_GEMM_DEFAULT_TENSOR_OP;
  } else {
    algo = HIPBLAS_GEMM_DEFAULT;
  }

  float alpha = 1.0f, beta_w = 1.0f, beta_x = 0.0f;
  // row-major
  if ((wgrad_[0])->get_format() == TensorFormat_t::HW &&
      in_tensor->get_format() == TensorFormat_t::HW &&
      out_tensor->get_format() == TensorFormat_t::HW) {
    // gradient respect to W
    CK_CUBLAS_THROW_(hipblasGemmEx(cublas_handle_, HIPBLAS_OP_N, HIPBLAS_OP_T, n, k, m, &alpha, out,
                                  HIP_R_32F, n, in, HIP_R_32F, k, &beta_w, wgrad, HIP_R_32F, n,
                                  HIP_R_32F, algo));
    // gradient respect to Xn
    CK_CUBLAS_THROW_(hipblasGemmEx(cublas_handle_, HIPBLAS_OP_T, HIPBLAS_OP_N, k, m, n, &alpha, weight,
                                  HIP_R_32F, n, out, HIP_R_32F, n, &beta_x, in, HIP_R_32F, k,
                                  HIP_R_32F, algo));
    cal_bias_grad(out, bias_grad, m, n, true, stream);
  }
  // Col-major
  else if ((weights_[0])->get_format() == TensorFormat_t::WH &&
           in_tensor->get_format() == TensorFormat_t::WH &&
           out_tensor->get_format() == TensorFormat_t::WH) {
    // gradient respect to W
    CK_CUBLAS_THROW_(hipblasGemmEx(cublas_handle_, HIPBLAS_OP_T, HIPBLAS_OP_N, k, n, m, &alpha, in,
                                  HIP_R_32F, m, out, HIP_R_32F, m, &beta_w, wgrad, HIP_R_32F, k,
                                  HIP_R_32F, algo));
    // gradient respect to Xn
    CK_CUBLAS_THROW_(hipblasGemmEx(cublas_handle_, HIPBLAS_OP_N, HIPBLAS_OP_T, m, k, n, &alpha, out,
                                  HIP_R_32F, m, weight, HIP_R_32F, k, &beta_x, in, HIP_R_32F, m,
                                  HIP_R_32F, algo));
    cal_bias_grad(out, bias_grad, m, n, false, stream);
  } else
    CK_THROW_(Error_t::UnSupportedFormat, "The format combination is not supported");
}

std::vector<float> FullyConnectedLayer::get_initializer() {
  std::vector<float> initializer;
  initializer.resize((weights_[0])->get_num_elements() + (weights_[1])->get_num_elements());
  const auto& in_tensor = in_tensors_[0];
  const auto& out_tensor = out_tensors_[0];
  float in_dim = in_tensor->get_format() == TensorFormat_t::WH ? (in_tensor->get_dims())[0]
                                                               : (in_tensor->get_dims())[1];
  float out_dim = out_tensor->get_format() == TensorFormat_t::WH ? (out_tensor->get_dims())[0]
                                                                 : (out_tensor->get_dims())[1];
  float limit = sqrt(6.f / (in_dim + out_dim));
  HugeCTR::UnifiedDataSimulator<float> fdata_sim(-1 * limit, limit);
  for (size_t i = 0; i < (weights_[0])->get_num_elements(); i++)
    initializer[i] = fdata_sim.get_num();
  for (size_t i = 0; i < (weights_[1])->get_num_elements(); i++)
    initializer[i + (weights_[0])->get_num_elements()] = 0.f;
  return initializer;
}

}  // namespace HugeCTR
